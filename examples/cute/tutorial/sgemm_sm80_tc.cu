#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2023 - 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: BSD-3-Clause
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * 1. Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * 2. Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * 3. Neither the name of the copyright holder nor the names of its
 * contributors may be used to endorse or promote products derived from
 * this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include <cstdlib>
#include <cstdio>
#include <cassert>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"
#include "cutlass/util/helper_cuda.hpp"
#include <cute/atom/mma_atom.hpp>

template <class ProblemShape, class CtaTiler,
          class TA, class AStride, class ASmemLayout, class TiledCopyA,
          class TB, class BStride, class BSmemLayout, class TiledCopyB,
          class TC, class CStride, class CSmemLayout, class TiledMma>
__global__ static
__launch_bounds__(decltype(size(TiledMma{}))::value)
void
gemm_device(ProblemShape shape_MNK, CtaTiler cta_tiler,
            TA const* A, AStride dA, ASmemLayout sA_layout, TiledCopyA copy_a,
            TB const* B, BStride dB, BSmemLayout sB_layout, TiledCopyB copy_b,
            TC      * C, CStride dC, CSmemLayout          , TiledMma mma)
{
  using namespace cute;

  // Preconditions
  CUTE_STATIC_ASSERT_V(rank(shape_MNK) == Int<3>{});                   // (M, N, K)
  CUTE_STATIC_ASSERT_V(rank(cta_tiler) == Int<3>{});                   // (BLK_M, BLK_N, BLK_K)

  CUTE_STATIC_ASSERT_V(size(copy_a) == size(mma));                     // NumThreads
  CUTE_STATIC_ASSERT_V(size(copy_b) == size(mma));                     // NumThreads

  static_assert(is_static<ASmemLayout>::value);
  static_assert(is_static<BSmemLayout>::value);
  static_assert(is_static<CSmemLayout>::value);

  CUTE_STATIC_ASSERT_V(size<0>(ASmemLayout{}) == size<0>(cta_tiler));  // BLK_M
  CUTE_STATIC_ASSERT_V(size<0>(CSmemLayout{}) == size<0>(cta_tiler));  // BLK_M
  CUTE_STATIC_ASSERT_V(size<0>(BSmemLayout{}) == size<1>(cta_tiler));  // BLK_N
  CUTE_STATIC_ASSERT_V(size<1>(CSmemLayout{}) == size<1>(cta_tiler));  // BLK_N
  CUTE_STATIC_ASSERT_V(size<1>(ASmemLayout{}) == size<2>(cta_tiler));  // BLK_K
  CUTE_STATIC_ASSERT_V(size<1>(BSmemLayout{}) == size<2>(cta_tiler));  // BLK_K

  CUTE_STATIC_ASSERT_V(congruent(select<0,2>(shape_MNK), dA));         // dA strides for shape MK
  CUTE_STATIC_ASSERT_V(congruent(select<1,2>(shape_MNK), dB));         // dB strides for shape NK
  CUTE_STATIC_ASSERT_V(congruent(select<0,1>(shape_MNK), dC));         // dC strides for shape MN

  //
  // Full and Tiled Tensors
  //

  // Represent the full tensors
  Tensor mA = make_tensor(make_gmem_ptr(A), select<0,2>(shape_MNK), dA); // (M,K)
  Tensor mB = make_tensor(make_gmem_ptr(B), select<1,2>(shape_MNK), dB); // (N,K)
  Tensor mC = make_tensor(make_gmem_ptr(C), select<0,1>(shape_MNK), dC); // (M,N)

  // Get the appropriate blocks for this thread block
  auto cta_coord = make_coord(blockIdx.x, blockIdx.y, _);              // (m,n,k)
  Tensor gA = local_tile(mA, cta_tiler, cta_coord, Step<_1, X,_1>{});  // (BLK_M,BLK_K,k)
  Tensor gB = local_tile(mB, cta_tiler, cta_coord, Step< X,_1,_1>{});  // (BLK_N,BLK_K,k)
  Tensor gC = local_tile(mC, cta_tiler, cta_coord, Step<_1,_1, X>{});  // (BLK_M,BLK_N)

  // Shared memory buffers
  __shared__ TA smemA[cosize_v<ASmemLayout>];
  __shared__ TB smemB[cosize_v<BSmemLayout>];
  Tensor sA = make_tensor(make_smem_ptr(smemA), sA_layout);            // (BLK_M,BLK_K,PIPE)
  Tensor sB = make_tensor(make_smem_ptr(smemB), sB_layout);            // (BLK_N,BLK_K,PIPE)

  auto mma_k = tile_size<2>(mma);
  auto sA_layout_p = flatten(logical_divide(sA_layout, make_tile(_, make_layout(mma_k), _)));  // (BLK_M, mma_k, BLK_mma_K, PIPE)
  auto sB_layout_p = flatten(logical_divide(sB_layout, make_tile(_, make_layout(mma_k), _)));  // (BLK_N, mma_k, BLK_mma_K, PIPE)
  CUTE_STATIC_ASSERT_V(rank(sA_layout_p) == Int<4>{});
  CUTE_STATIC_ASSERT_V(rank(sB_layout_p) == Int<4>{});
  CUTE_STATIC_ASSERT_V(size<1>(sA_layout_p) == mma_k);
  CUTE_STATIC_ASSERT_V(size<1>(sB_layout_p) == mma_k);
  Tensor sA_p = make_tensor(make_smem_ptr(smemA), sA_layout_p);        // (BLK_M, mma_k, BLK_mma_K, PIPE)
  Tensor sB_p = make_tensor(make_smem_ptr(smemB), sB_layout_p);        // (BLK_N, mma_k, BLK_mma_K, PIPE)

  //
  // Partition the copying of A and B tiles across the threads
  //

  ThrCopy thr_copy_a = copy_a.get_slice(threadIdx.x);
  Tensor tAgA = thr_copy_a.partition_S(gA);                            // (CPY,CPY_M,CPY_K,k)
  Tensor tAsA = thr_copy_a.partition_D(sA);                            // (CPY,CPY_M,CPY_K,PIPE)

  ThrCopy thr_copy_b = copy_b.get_slice(threadIdx.x);
  Tensor tBgB = thr_copy_b.partition_S(gB);                            // (CPY,CPY_N,CPY_K,k)
  Tensor tBsB = thr_copy_b.partition_D(sB);                            // (CPY,CPY_N,CPY_K,PIPE)

  CUTE_STATIC_ASSERT_V(size<1>(tAgA) == size<1>(tAsA));                // CPY_M
  CUTE_STATIC_ASSERT_V(size<2>(tAgA) == size<2>(tAsA));                // CPY_K
  CUTE_STATIC_ASSERT_V(size<1>(tBgB) == size<1>(tBsB));                // CPY_N
  CUTE_STATIC_ASSERT_V(size<2>(tBgB) == size<2>(tBsB));                // CPY_K

  //
  // PREFETCH
  //

  auto K_PIPE_MAX = size<3>(tAsA);

  // Total count of tiles
  int k_tile_count = size<3>(tAgA);
  // Current tile index in gmem to read from
  int k_tile_next = 0;

  // Start async loads for all pipes but the last
  CUTE_UNROLL
  for (int k_pipe = 0; k_pipe < K_PIPE_MAX-1; ++k_pipe) {
    copy(copy_a, tAgA(_,_,_,k_tile_next), tAsA(_,_,_,k_pipe));
    copy(copy_b, tBgB(_,_,_,k_tile_next), tBsB(_,_,_,k_pipe));
    cp_async_fence();
    --k_tile_count;
    if (k_tile_count > 0) { ++k_tile_next; }
  }

  //
  // Define A/B partitioning and C accumulators
  //

  ThrMMA thr_mma = mma.get_slice(threadIdx.x);
  Tensor tCsA = thr_mma.partition_A(sA_p);                             // (MMA,MMA_M,MMA_K,BLK_mma_K,PIPE)
  Tensor tCsB = thr_mma.partition_B(sB_p);                             // (MMA,MMA_N,MMA_K,BLK_mma_K,PIPE)
  Tensor tCgC = thr_mma.partition_C(gC);                               // (MMA,MMA_M,MMA_N)

  CUTE_STATIC_ASSERT_V((size<3>(tCsA) == size<2>(sA_p)));
  CUTE_STATIC_ASSERT_V((size<3>(tCsB) == size<2>(sB_p)));

  // Allocate registers for pipelining
  Tensor tCrA = thr_mma.make_fragment_A(tCsA(_,_,_,_,0));                // (MMA,MMA_M,MMA_K,BLK_mma_K)
  Tensor tCrB = thr_mma.make_fragment_B(tCsB(_,_,_,_,0));                // (MMA,MMA_N,MMA_K,BLK_mma_K)
  // Allocate the accumulators -- same size as the projected data
  Tensor tCrC = thr_mma.make_fragment_C(tCgC);                         // (MMA,MMA_M,MMA_N)

  CUTE_STATIC_ASSERT_V((  shape(tCrA) == take<0,4>(shape(tCsA))));     // (MMA,MMA_M,MMA_K)
  CUTE_STATIC_ASSERT_V((  shape(tCrB) == take<0,4>(shape(tCsB))));     // (MMA,MMA_N,MMA_K)
  CUTE_STATIC_ASSERT_V((  shape(tCrC) == take<0,3>(shape(tCgC))));     // (MMA,MMA_M,MMA_N)
  CUTE_STATIC_ASSERT_V((size<1>(tCgC) == size<1>(tCsA)));              // MMA_M
  CUTE_STATIC_ASSERT_V((size<2>(tCgC) == size<1>(tCsB)));              // MMA_N
  CUTE_STATIC_ASSERT_V((size<2>(tCsA) == size<2>(tCsB)));              // MMA_K

  // Clear the accumulators
  clear(tCrC);

#if 0
  if(thread0()) {
    print("  mA : "); print(  mA); print("\n");
    print("  gA : "); print(  gA); print("\n");
    print("  sA : "); print(  sA); print("\n");
    print("tAgA : "); print(tAgA); print("\n");
    print("tAsA : "); print(tAsA); print("\n");
  }
#endif

#if 0
  if(thread0()) {
    print("  mB : "); print(  mB); print("\n");
    print("  gB : "); print(  gB); print("\n");
    print("  sB : "); print(  sB); print("\n");
    print("tBgB : "); print(tBgB); print("\n");
    print("tBsB : "); print(tBsB); print("\n");
  }
#endif

#if 0
  if(thread0()) {
    print("  mC : "); print(  mC); print("\n");
    print("  gC : "); print(  gC); print("\n");
    print("tCsA : "); print(tCsA); print("\n");
    print("tCsB : "); print(tCsB); print("\n");
    print("tCgC : "); print(tCgC); print("\n");
    print("tCrA : "); print(tCrA); print("\n");
    print("tCrB : "); print(tCrB); print("\n");
    print("tCrC : "); print(tCrC); print("\n");
  }
#endif

#if 1

  // Current pipe index in smem to read from
  int smem_pipe_read  = 0;
  // Current pipe index in smem to write to
  int smem_pipe_write = K_PIPE_MAX-1;

  Tensor tCsA_p = tCsA(_,_,_,_,smem_pipe_read);
  Tensor tCsB_p = tCsB(_,_,_,_,smem_pipe_read);

  // Size of the register pipeline
  auto K_BLOCK_MAX = size<3>(tCrA);

  // PREFETCH register pipeline
  if (K_BLOCK_MAX > 1) {
    // Wait util our first prefetched tile is loaded in
    cp_async_wait<K_PIPE_MAX-2>();
    __syncthreads();

    // Prefetch the first rmem from the first k-tile
    copy(tCsA_p(_,_,_,Int<0>{}), tCrA(_,_,_,Int<0>{}));
    copy(tCsB_p(_,_,_,Int<0>{}), tCrB(_,_,_,Int<0>{}));
  }

  // Don't need the register pipeline with the use of tensor cores
  CUTE_NO_UNROLL
  while (k_tile_count > -(K_PIPE_MAX - 1))
  {
    CUTE_UNROLL
    for (int k_block = 0; k_block < K_BLOCK_MAX; ++k_block) {
      if (k_block == K_BLOCK_MAX - 1) {
        // Slice the smem_pipe_read smem
        tCsA_p = tCsA(_,_,_,_,smem_pipe_read);
        tCsB_p = tCsB(_,_,_,_,smem_pipe_read);

        // Commit the smem for smem_pipe_read
        cp_async_wait<K_PIPE_MAX-2>();
        __syncthreads();
      }

      // Load A, B shmem->regs for k_block+1
      auto k_block_next = (k_block + Int<1>{}) % K_BLOCK_MAX;      // static
      copy(tCsA_p(_,_,_,k_block_next), tCrA(_,_,_,k_block_next));
      copy(tCsB_p(_,_,_,k_block_next), tCrB(_,_,_,k_block_next));

      // Copy gmem to smem before computing gemm on each k-pipe
      if (k_block == 0) {
        copy(copy_a, tAgA(_,_,_,k_tile_next), tAsA(_,_,_,smem_pipe_write));
        copy(copy_b, tBgB(_,_,_,k_tile_next), tBsB(_,_,_,smem_pipe_write));
        cp_async_fence();

        // Advance the gmem tile
        --k_tile_count;
        if (k_tile_count > 0) { ++k_tile_next; }

        // Advance the smem pipe
        smem_pipe_write = smem_pipe_read;
        ++smem_pipe_read;
        smem_pipe_read = (smem_pipe_read == K_PIPE_MAX) ? 0 : smem_pipe_read;
      }
      gemm(mma, tCrA(_,_,_,k_block), tCrB(_,_,_,k_block), tCrC);
    }
  }
#endif

  //
  // Epilogue
  //

  copy(tCrC, tCgC);

  // axpby(alpha, tCrC, beta, tCgC);
}

// Setup params for a NT GEMM
template <class TA, class TB, class TC>
void
gemm_nt(int m, int n, int k,
        TA const* A, int ldA,
        TB const* B, int ldB,
        TC      * C, int ldC,
        hipStream_t stream = 0)
{
  using namespace cute;

  // Define shapes (dynamic)
  auto M = int(m);
  auto N = int(n);
  auto K = int(k);
  auto prob_shape = make_shape(M, N, K);                     // (M, N, K)

  // Define NT strides (mixed)
  auto dA = make_stride(Int<1>{}, ldA);                      // (dM, dK)
  auto dB = make_stride(Int<1>{}, ldB);                      // (dN, dK)
  auto dC = make_stride(Int<1>{}, ldC);                      // (dM, dN)

  // Define CTA tile sizes (static)
  auto bM = Int<128>{};
  auto bN = Int<128>{};
  auto bK = Int< 16>{};
  auto cta_tiler = make_shape(bM, bN, bK);                   // (BLK_M, BLK_N, BLK_K)
  auto bP = Int<4>{};  // Pipeline

  // Define the smem layouts (static)
  auto sA = make_layout(make_shape(bM, bK, bP));             // (m,k,p) -> smem_idx; m-major
  auto sB = make_layout(make_shape(bN, bK, bP));             // (n,k,p) -> smem_idx; n-major
  auto sC = make_layout(make_shape(bM, bN));                 // (m,n) -> smem_idx; m-major

  // Define the thread layouts (static)

  TiledCopy copyA = make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, TA>{},
                                    Layout<Shape<_16,_16>>{}, // Thr layout 32x8 m-major
                                    Layout<Shape< _8,_1>>{});// Val layout  4x1 m-major
  TiledCopy copyB = make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, TB>{},
                                    Layout<Shape<_16,_16>>{}, // Thr layout 32x8 n-major
                                    Layout<Shape< _8,_1>>{});// Val layout  4x1 n-major
  // SM80_16x8x8_F16F16F16F16_TN
  TiledMMA mmaC = make_tiled_mma(SM80_16x8x8_F16F16F16F16_TN{},
                                 Layout<Shape<_2,_4>>{});  // 16x8x8 TiledMMA

#if 0
  print(copyA);
  print(copyB);
  print(mmaC);
#endif

#if 0
  print_latex(copyA);
  print_latex(copyB);
  print_latex(mmaC);
#endif

  dim3 dimBlock(size(mmaC));
  dim3 dimGrid(size(ceil_div(M, bM)),
               size(ceil_div(N, bN)));
  gemm_device<<<dimGrid, dimBlock, 0, stream>>>
      (prob_shape, cta_tiler,
       A, dA, sA, copyA,
       B, dB, sB, copyB,
       C, dC, sC, mmaC);
}

// Setup params for a TN GEMM, K-Major inputs
template <class TA, class TB, class TC>
void
gemm_tn(int m, int n, int k,
        TA const* A, int ldA,
        TB const* B, int ldB,
        TC      * C, int ldC,
        hipStream_t stream = 0)
{
  using namespace cute;

  // Define shapes (dynamic)
  auto M = int(m);
  auto N = int(n);
  auto K = int(k);
  auto prob_shape = make_shape(M, N, K);                     // (M, N, K)

  // Define TN strides (mixed)
  auto dA = make_stride(ldA, Int<1>{});                      // (dM, dK)
  auto dB = make_stride(ldB, Int<1>{});                      // (dN, dK)
  auto dC = make_stride(ldC, Int<1>{});                      // (dM, dN)

  // Define CTA tile sizes (static)
  auto bM = Int<128>{};
  auto bN = Int<128>{};
  auto bK = Int< 16>{};
  auto cta_tiler = make_shape(bM, bN, bK);                   // (BLK_M, BLK_N, BLK_K)
  auto bP = Int<4>{};  // Pipeline

  // Define the smem layouts (static)
  auto sA_atom = make_layout(make_shape (bM, bK), LayoutRight{}); // (m,k) -> smem_idx; padded k-major
  auto sB_atom = make_layout(make_shape (bN, bK), LayoutRight{}); // (n,k) -> smem_idx; padded k-major
  auto sA = tile_to_shape(sA_atom, make_shape(bM, bK, bP));
  auto sB = tile_to_shape(sB_atom, make_shape(bN, bK, bP));
  auto sC = make_layout(make_shape(bM, bN));                        // (m,n) -> smem_idx

  // Define the thread layouts (static)
  TiledCopy copyA = make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, TA>{},
                                    make_layout(Shape<_128,_2>{}, LayoutRight{}),
                                    Layout<Shape< _1,_8>>{});
                                    
  TiledCopy copyB = make_tiled_copy(Copy_Atom<SM80_CP_ASYNC_CACHEALWAYS<uint128_t>, TB>{},
                                    make_layout(Shape<_128,_2>{}, LayoutRight{}),
                                    Layout<Shape< _1,_8>>{});
  // SM80_16x8x8_F16F16F16F16_TN
  TiledMMA mmaC = make_tiled_mma(SM80_16x8x8_F16F16F16F16_TN{},
                                 Layout<Shape<_2,_4>>{});  // 16x8x8 TiledMMA

#if 0
  print(copyA);
  print(copyB);
  print(mmaC);
#endif

#if 0
  print_latex(copyA);
  print_latex(copyB);
  print_latex(mmaC);
#endif

  dim3 dimBlock(size(mmaC));
  dim3 dimGrid(size(ceil_div(M, bM)),
               size(ceil_div(N, bN)));
  gemm_device<<<dimGrid, dimBlock, 0, stream>>>
      (prob_shape, cta_tiler,
       A, dA, sA, copyA,
       B, dB, sB, copyB,
       C, dC, sC, mmaC);
}


template <class TA, class TB, class TC>
void
gemm(char transA, char transB, int m, int n, int k,
     TA const* A, int ldA,
     TB const* B, int ldB,
     TC      * C, int ldC,
     hipStream_t stream = 0)
{
  // if (transA == 'N' && transB == 'T') {
  //   return gemm_nt(m, n, k, A, ldA, B, ldB, C, ldC, stream);
  // }
  if (transA == 'T' && transB == 'N') {
    return gemm_tn(m, n, k, A, ldA, B, ldB, C, ldC, stream);
  }
  assert(false && "Not implemented");
}


int main(int argc, char** argv)
{
  hipDeviceProp_t props;
  hipError_t error = hipGetDeviceProperties(&props, 0);
  if (error != hipSuccess) {
    std::cerr << "hipGetDeviceProperties() returned an error: " << hipGetErrorString(error) << std::endl;
    return -1;
  }

  if (props.major < 8) { 
    std::cout << "This example requires an Ampere GPU or newer (CC >= 80)" << std::endl;
    // Return 0 so tests pass if run on unsupported architectures or CUDA Toolkits.
    return 0;
  }

  int m = 8192;
  if (argc >= 2)
    sscanf(argv[1], "%d", &m);

  int n = 8192;
  if (argc >= 3)
    sscanf(argv[2], "%d", &n);

  int k = 4096;
  if (argc >= 4)
    sscanf(argv[3], "%d", &k);

  char transA = 'T';
  if (argc >= 5)
    sscanf(argv[4], "%c", &transA);

  char transB = 'N';
  if (argc >= 6)
    sscanf(argv[5], "%c", &transB);

  using TA = half;
  using TB = half;
  using TC = half;

  std::cout << "M = " << m << std::endl;
  std::cout << "N = " << n << std::endl;
  std::cout << "K = " << k << std::endl;
  std::cout << "C = A^" << transA << " B^" << transB << std::endl;

  thrust::host_vector<TA> h_A(m*k);
  thrust::host_vector<TB> h_B(n*k);
  thrust::host_vector<TC> h_C(m*n);

  for (int j = 0; j < m*k; ++j) h_A[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < n*k; ++j) h_B[j] = static_cast<TB>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < m*n; ++j) h_C[j] = static_cast<TC>(-1);

  thrust::device_vector<TA> d_A = h_A;
  thrust::device_vector<TB> d_B = h_B;
  thrust::device_vector<TC> d_C = h_C;

  double gflops = (2.0*m*n*k) * 1e-9;

  const int timing_iterations = 100;
  GPU_Clock timer;

  int ldA = 0, ldB = 0, ldC = m;

  if (transA == 'N') {
    ldA = m;
  } else if (transA == 'T') {
    ldA = k;
  } else {
    assert(false);
  }

  if (transB == 'N') {
    ldB = k;
  } else if (transB == 'T') {
    ldB = n;
  } else {
    assert(false);
  }

  // Run once
  d_C = h_C;
  gemm(transA, transB, m, n, k,
       d_A.data().get(), ldA,
       d_B.data().get(), ldB,
       d_C.data().get(), ldC);
  CUTE_CHECK_LAST();
  thrust::host_vector<TC> cute_result = d_C;

  // Timing iterations
  timer.start();
  for (int i = 0; i < timing_iterations; ++i) {
    gemm(transA, transB, m, n, k,
         d_A.data().get(), ldA,
         d_B.data().get(), ldB,
         d_C.data().get(), ldC);
  }
  double cute_time = timer.seconds() / timing_iterations;
  CUTE_CHECK_LAST();
  printf("CUTE_GEMM:     [%6.1f]GFlop/s  (%6.4f)ms\n", gflops / cute_time, cute_time*1000);

  return 0;
}
